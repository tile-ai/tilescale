#include <torch/extension.h>

#include <iostream>

#include "cute/tensor.hpp"
#include "cutlass/cutlass.h"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/transform/device/transform_universal_adapter.hpp"
#include "cutlass/transform/kernel/sparse_gemm_compressor.hpp"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"

using namespace cute;

#define CUTLASS_CHECK(status)                                             \
  {                                                                       \
    cutlass::Status error = status;                                       \
    if (error != cutlass::Status::kSuccess) {                             \
      std::cerr << "Got cutlass error: " << cutlassGetStatusString(error) \
                << " at: " << __LINE__ << std::endl;                      \
      exit(EXIT_FAILURE);                                                 \
    }                                                                     \
  }

#define CUDA_CHECK(status)                                              \
  {                                                                     \
    hipError_t error = status;                                         \
    if (error != hipSuccess) {                                         \
      std::cerr << "Got bad cuda status: " << hipGetErrorString(error) \
                << " at line: " << __LINE__ << std::endl;               \
      exit(EXIT_FAILURE);                                               \
    }                                                                   \
  }
template<typename T, int BlockK, bool transposed>
std::tuple<torch::Tensor, torch::Tensor> compress_impl(torch::Tensor A) {
  using ElementA = T;
  using ElementE = uint8_t;
  using LayoutTagA = conditional_t<transposed, cutlass::layout::ColumnMajor, cutlass::layout::RowMajor>;
  using ProblemShape = cute::Shape<int, int, int, int>;

  using StrideA = cutlass::gemm::TagToStrideA_t<LayoutTagA>;
  using StrideE = StrideA;

  // NOTE: this is derived from sparse sm90 mma atoms
  // Ref: https://github.com/NVIDIA/cutlass/blob/dc4817921edda44a549197ff3a9dcf5df0636e7b/include/cute/atom/mma_traits_sm90_gmma_sparse.hpp
  using SparseE = conditional_t<(sizeof_bits_v<ElementA> == 32), cute::sparse_elem<4, ElementE>, cute::sparse_elem<8, ElementE>>;
  static constexpr GMMA::Major GmmaMajorA = transposed ? cute::SM90::GMMA::Major::MN : cute::SM90::GMMA::Major::K;
  using SparseConfig = cutlass::Sm90GemmSparseConfig<
      cute::sparse_elem<2, ElementA>, GmmaMajorA,
      SparseE, cute::C<BlockK>>;

  using CompressorUtility =
      cutlass::transform::kernel::StructuredSparseCompressorUtility<
          ProblemShape, ElementA, LayoutTagA, SparseConfig>;

  using CompressorKernel = cutlass::transform::kernel::StructuredSparseCompressor<
      ProblemShape, ElementA, LayoutTagA, SparseConfig, cutlass::arch::Sm90>;

  using Compressor = cutlass::transform::device::TransformUniversalAdapter<CompressorKernel>;

  TORCH_CHECK(A.is_contiguous(), "A need to be contiguous");
  TORCH_CHECK(A.dim() == 2, "Might support batch dim in the future ");

  int M = -1;
  int K = -1;
  int N = -1;  // not used, but required for config
  int L = 1;
  if constexpr(transposed) {
    M = A.size(1);
    K = A.size(0);
  } else {
    M = A.size(0);
    K = A.size(1);
  }

  ProblemShape problem_shape = make_tuple(M, N, K, L);
  StrideA stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));

  CompressorUtility compressor_utility(problem_shape, stride_A);
  int ME = compressor_utility.get_metadata_m_physical();
  int KE = compressor_utility.get_metadata_k_physical();
  int KC = compressor_utility.get_tensorA_k_physical();

  StrideE stride_E = cutlass::make_cute_packed_stride(StrideE{}, cute::make_shape(ME, KE, L));
  auto dtype = A.dtype().toScalarType();
  torch::Tensor A_compressed = torch::zeros(KC * M,
        torch::TensorOptions().dtype(dtype).device(A.device()));
  torch::Tensor E = torch::zeros({ME, KE},
      torch::TensorOptions().dtype(torch::kUInt8).device(A.device()));

  cutlass::KernelHardwareInfo hw_info;
  hw_info.device_id = A.device().index();
  hw_info.sm_count =
      cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  typename Compressor::Arguments arguments{problem_shape,
                                           {
                                               A.data_ptr(),
                                               stride_A,
                                               A_compressed.data_ptr(),
                                               E.data_ptr(),
                                           },
                                           {hw_info}};

  Compressor compressor_op;
  size_t workspace_size = Compressor::get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  CUTLASS_CHECK(compressor_op.can_implement(arguments));
  CUTLASS_CHECK(compressor_op.initialize(arguments, workspace.get()));
  CUTLASS_CHECK(compressor_op.run());
  CUDA_CHECK(hipDeviceSynchronize());

  if constexpr (transposed) {
    return std::make_tuple(A_compressed.view({KC, M}), E);
  } else {
    return std::make_tuple(A_compressed.view({M, KC}), E);
  }
}

// block <= 128
// Ref https://github.com/NVIDIA/cutlass/blob/c2ad7c5b20f131c4ba33601860f1da3f9c9df0f3/include/cutlass/gemm/collective/builders/sm90_sparse_gmma_builder.inl#L145-L146
#define DISPATCH_BLOCK_K(TYPE, BLOCK_K, FACTOR, TENSOR, TRANSPOSED)                                        \
  [&]() -> std::tuple<torch::Tensor, torch::Tensor> {                                                      \
    switch (BLOCK_K) {                                                                                     \
      case int(32 * FACTOR): return compress_impl<TYPE, int(32 * FACTOR), TRANSPOSED>(TENSOR);             \
      case int(64 * FACTOR): return compress_impl<TYPE, int(64 * FACTOR), TRANSPOSED>(TENSOR);             \
      case int(128 * FACTOR): return compress_impl<TYPE, int(128 * FACTOR), TRANSPOSED>(TENSOR);           \
      default:                                                                                             \
        TORCH_CHECK(false, "Unsupported block_k: ", BLOCK_K);                                              \
    }                                                                                                      \
  }()

#define DISPATCH_CONTIGUOUS(TRANSPOSED)                                                                    \
  [&]() -> std::tuple<torch::Tensor, torch::Tensor> {                                                      \
    switch (dtype) {                                                                                       \
      case torch::kFloat32:                                                                                \
        return DISPATCH_BLOCK_K(float, block_k, 0.5, A, TRANSPOSED);                                       \
      case torch::kFloat16:                                                                                \
      case torch::kBFloat16:                                                                               \
        return DISPATCH_BLOCK_K(cute::half_t, block_k, 1, A, TRANSPOSED);                                  \
      case torch::kFloat8_e4m3fn:                                                                          \
        return DISPATCH_BLOCK_K(cute::float_e4m3_t, block_k, 2, A, TRANSPOSED);                            \
      case torch::kFloat8_e5m2:                                                                            \
        return DISPATCH_BLOCK_K(cute::float_e5m2_t, block_k, 2, A, TRANSPOSED);                            \
      case torch::kChar:                                                                                   \
        return DISPATCH_BLOCK_K(int8_t, block_k, 2, A, TRANSPOSED);                                        \
      case torch::kByte:                                                                                   \
        return DISPATCH_BLOCK_K(uint8_t, block_k, 2, A, TRANSPOSED);                                       \
      default:                                                                                             \
        TORCH_CHECK(false, "Unsupported dtype");                                                           \
    }                                                                                                      \
  }()

std::tuple<torch::Tensor, torch::Tensor> compress_sm90(torch::Tensor A, int64_t block_k, bool transposed) {
  auto dtype = A.dtype().toScalarType();
  return transposed ? DISPATCH_CONTIGUOUS(true) : DISPATCH_CONTIGUOUS(false);
}

#undef DISPATCH_BLOCK_K
#undef DISPATCH_CONTIGUOUS

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("compress_sm90", torch::wrap_pybind_function(compress_sm90),
        "compress_sm90");
}
